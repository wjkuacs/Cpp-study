#include <iostream>
#include <hip/hip_runtime.h>

// 定义向量长度
#define N 1024

// 定义一个 CUDA 内核函数：向量加法
__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements)
{
    // 每个线程计算一个元素
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    // 主机端数据
    float h_A[N], h_B[N], h_C[N];
    
    // 初始化数据
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // 设备端数据
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // 将数据复制到设备
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // 启动CUDA内核
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 将结果复制回主机
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // 验证结果
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            success = false;
            break;
        }
    }

    if (success) {
        std::cout << "Vector addition successful!" << std::endl;
    } else {
        std::cout << "Vector addition failed!" << std::endl;
    }

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}